#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include "stdio.h"
#include "stdlib.h"
#include "omp.h"

#define iter 1000
__global__ void JacobiGPU(double* A, double* F, double* x_now, double* x_next, int N) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        double sigma = 0.0;
        int idx_Ai = idx * N;
        for (int i = 0; i < N; i++) {
            if (idx != i) sigma += A[idx_Ai + i] * x_now[i];

        }
        x_next[idx] = (F[idx] - sigma) / A[idx_Ai + idx];

    }
}

void Jacobi(int N, double** A, double* F, double* x_now, double* x_next)
{
    int i, j;
    double sigma;

    for (i = 0; i < N; i++)
    {
        sigma = 0.0;
        for (j = 0; j < N; j++)
        {
            if (i != j)
                sigma += A[i][j] * x_now[j];
        }
        x_next[i] = (F[i] - sigma) / A[i][i];
    }
}
int main(int argc, char** argv)
{
    char* filename = "input.txt";
    /*if (argv[1] != NULL) {
        filename = argv[1];
    }
    else {
        printf("Text file needed\n");
        return 0;
    }*/

    FILE* file = fopen(filename, "r");
    if (file == NULL) {
        perror("Couldn't open the input file");
    }

    int n;
    fscanf(file, "%d", &n);

    double** A = (double**)malloc(n * sizeof(double*));
    for (int i = 0; i < n; i++)
        A[i] = (double*)malloc(n * sizeof(double));
    double* F = (double*)malloc(n * sizeof(double));
    double* X = (double*)malloc(n * sizeof(double));
    double* T = (double*)malloc(n * sizeof(double));

    for (unsigned long int i = 0; i < n; ++i){
        for (unsigned long int j = 0; j < n; ++j){
            
            fscanf(file, "%lf", A[i] + j);
            if (i != j)
                A[i][j] /= 10000;
        }

        fscanf(file, "%lf\n", F + i);
        T[i] = 0;
        X[i] = 0;
    }

    fclose(file);

    double* newAr;
    newAr = (double*)malloc(n * n * sizeof(double));
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {

            newAr[i * n + j] = A[i][j];
        }
    }

    double* X_GPU, * T_GPU, * A_GPU, * F_GPU;

    hipMalloc((void**)&X_GPU, n * sizeof(double));
    hipMalloc((void**)&T_GPU, n * sizeof(double));
    hipMalloc((void**)&A_GPU, n * n * sizeof(double));
    hipMalloc((void**)&F_GPU, n * sizeof(double));

    hipMemcpy(T_GPU, T, sizeof(double) * n, hipMemcpyHostToDevice);
    hipMemcpy(A_GPU, newAr, sizeof(double) * n * n, hipMemcpyHostToDevice);
    hipMemcpy(X_GPU, X, sizeof(double) * n, hipMemcpyHostToDevice);
    hipMemcpy(F_GPU, F, sizeof(double) * n, hipMemcpyHostToDevice);
    
    double timeStart = omp_get_wtime();
    for (int k = 0; k < iter; k++) {
        if (k % 2)
            Jacobi(n, A, F, X, T);
        else
            Jacobi(n, A, F, T, X);
    }

    //for (int i = 0; i < n; i++)
        //printf("%lf", X[i]);

    double workingTime = omp_get_wtime() - timeStart;
    printf("Algorithm done, time: %lf\n", workingTime);
    int tileSize = 1024;
    printf("Starting GPU algorithm with %d Tiles and %d Tile size\n", 1, tileSize);
    timeStart = omp_get_wtime();
    for (int k = 0; k < iter; k++) {
        if (k % 2)
            JacobiGPU << < 1, tileSize >> > (A_GPU, F_GPU, X_GPU, T_GPU, n);

        else
            JacobiGPU << < 1, tileSize >> > (A_GPU, F_GPU, T_GPU, X_GPU, n);
    }
    workingTime = omp_get_wtime() - timeStart;
    printf("Algorithm done, time: %lf\n", workingTime);
    if (n < 50) {
        printf("Printing results\n");
        FILE* CPUres, * GPUres;
        GPUres = fopen("GPUres.txt", "w+");
        CPUres = fopen("CPUres.txt", "w+");
        if (GPUres == NULL || CPUres == NULL) {
            printf("Can`t print results\n");
            return 0;
        }
        double* outGPU;
        outGPU = (double*)malloc(n * sizeof(double));
        hipMemcpy(outGPU, T_GPU, sizeof(double) * n, hipMemcpyDeviceToHost);
        for (int i = 0; i < n; i++) {
            fprintf(CPUres, "%lf\n", T[i]);
            fprintf(GPUres, "%lf\n", outGPU[i]);
        }
        fclose(GPUres); fclose(CPUres);
        for (int i = 0; i < n; i++) {
            if (T[i] != outGPU[i])
            {
                printf("Values are not equal\n");
                return 0;
            }

        }
        printf("Values are equal\n");

    }
    for (int i = 0; i < n; i++)
        free(A[i]);
    free(F); free(X); free(T); free(newAr); free(A);
    hipFree(X_GPU); hipFree(T_GPU); hipFree(A_GPU); hipFree(F_GPU);
    return 0;
}
